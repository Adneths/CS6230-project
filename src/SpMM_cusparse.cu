#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""
#include <hipsparse.h>

#include "SpMM_cusparse.h"

#ifdef PROFILE
#include "timer.h"
#endif

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return nullptr;                                                        \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return nullptr;                                                        \
    }                                                                          \
}

namespace cusparse {

Matrix<double>* spmm(CSRMatrix<double>* A, Matrix<double>* B) {
#ifdef PROFILE
    Timer timer;
    auto time = timer.tick();
#endif
    double              alpha       = 1.0f;
    double              beta        = 0.0f;
    hipsparseOperation_t opA         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipDataType        computeType = HIP_R_64F;

    // Allocate
    int *d_rowPtrA, *d_dataColA;
    double *d_dataValA, *d_B, *d_C;
    size_t
    rowPtrA_size  = (A->rows+1) * sizeof(int),
    dataColA_size = (A->nnz) * sizeof(int),
    dataValA_size = (A->nnz) * sizeof(double),
    B_size        = B->rows * B->cols;
    C_size        = A->rows * B->cols;
    
    hipMalloc(&d_rowPtrA, rowPtrA_size);
    hipMalloc(&d_dataColA, dataColA_size);
    hipMalloc(&d_dataValA, dataValA_size);
    hipMalloc(&d_B, B_size);
    hipMalloc(&d_C, C_size);

    // Copy
    hipMemcpy(d_rowPtrA , A->rowPtr , rowPtrA_size , hipMemcpyHostToDevice);
    hipMemcpy(d_dataColA, A->dataCol, dataColA_size, hipMemcpyHostToDevice);
    hipMemcpy(d_dataValA, A->dataVal, dataValA_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B , B->data , B_size , hipMemcpyHostToDevice);
    //--------------------------------------------------------------------------
    // Cusparse Setup
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A->rows, A->cols, A->nnz,
                                      d_rowPtrA, d_dataColA, d_dataValA,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) )
    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, B->rows, B->cols, B->rows, d_B,
                                        HIP_R_64F, B->order==ROW_MAJOR ? HIPSPARSE_ORDER_ROW : HIPSPARSE_ORDER_COL) )
    // Create dense matrix C
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, A->rows, B->cols, A->rows, d_C,
                                        HIP_R_64F, B->order==ROW_MAJOR ? HIPSPARSE_ORDER_ROW : HIPSPARSE_ORDER_COL) )
    //--------------------------------------------------------------------------
#ifdef PROFILE
    time = timer.tick();
    std::cout << "CuSparse Setup: " << time << std::endl;
    timer.tick();
#endif
    // SpMM Computation
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_64F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute SpMM
    CHECK_CUSPARSE( hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_64F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )
#ifdef PROFILE
    time = timer.tick();
    std::cout << "CuSparse Compute: " << time << std::endl;
    timer.tick();
#endif
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    
    double *h_C;
    h_C = (double*)malloc(A->rows * B->cols * sizeof(double));

    hipMemcpy(h_C, d_C, A->rows * B->cols * sizeof(double), hipMemcpyDeviceToHost);

    Matrix<double>* ret = new Matrix<double>(A->rows, B->cols, h_C, B->order);

    hipFree(dBuffer);
    hipFree(d_rowPtrA);
    hipFree(d_dataColA);
    hipFree(d_dataValA);
    hipFree(d_B);
    hipFree(d_C);
#ifdef PROFILE
    time = timer.tick();
    std::cout << "CuSparse Teardown: " << time << std::endl;
#endif
    
    return ret;
}
}
