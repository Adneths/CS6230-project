#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""

#include "SpGEMM_cuda.h"

#ifdef PROFILE
#include "timer.h"
#endif

namespace cuda
{

    // Sparse General Matrix Multiplication with Dense Accumulator
    __global__ void spmv_kx(int *rowPtrA, int *dataColA, double *dataValA,
                            double *dense_vector,
                            double *dataValC)
    {
        int tx = threadIdx.x;
        int bx = blockIdx.x;
        if (tx < colsB)
        {
            int as = rowPtrA[bx];
            int ae = rowPtrA[bx + 1];
            for (int i = as; i < ae; i++)
            {
                double valA = dataValA[i];
                int c = dataColA[i];

                int bs = rowPtrB[c];
                int be = rowPtrB[c + 1];
                if (bs + tx < be)
                {
                    dataValC[bx * colsB + dataColB[bs + tx]] += valA * dataValB[bs + tx];
                }
            }
        }
    }

    CSRMatrix<double> *spmv(CSRMatrix<double> *A, double *DenseVectorB)
    {
#ifdef PROFILE
        Timer timer;
        auto time = timer.tick();
#endif
        int *d_rowPtrA, *d_dataColA;
        double *d_dataValA, *VectorOnGpu, *d_dataValC, *h_dataValC;

        size_t
            rowPtrA_size = (A->rows + 1) * sizeof(int),
            dataColA_size = (A->nnz) * sizeof(int),
            dataValA_size = (A->nnz) * sizeof(double),
            DenseVectorB_size = (DenseVectorB->row_num) * sizeof(double),
            dataValC_size = (A->rows * DenseVectorB->cols) * sizeof(double);

        hipMalloc(&d_rowPtrA, rowPtrA_size);
        hipMalloc(&d_dataColA, dataColA_size);
        hipMalloc(&d_dataValA, dataValA_size);
        hipMalloc(&VectorOnGpu, DenseVectorB_size);
        hipMalloc(&d_dataValC, dataValC_size);

        hipMemcpy(d_rowPtrA, A->rowPtr, rowPtrA_size, hipMemcpyHostToDevice);
        hipMemcpy(d_dataColA, A->dataCol, dataColA_size, hipMemcpyHostToDevice);
        hipMemcpy(d_dataValA, A->dataVal, dataValA_size, hipMemcpyHostToDevice);
        hipMemcpy(VectorOnGpu, DenseVectorB, DenseVectorB_size, hipMemcpyHostToDevice);
        hipMemset(d_dataValC, 0, dataValC_size);

        dim3 threadsPerBlock(32 * ((B->cols + 31) / 32));
        dim3 numBlocks(A->rows);
#ifdef PROFILE
        time = timer.tick();
        std::cout << "Cuda Setup: " << time << std::endl;
        timer.tick();
#endif

        spmv_kx<<<numBlocks, threadsPerBlock>>>(d_rowPtrA, d_dataColA, d_dataValA,
                                                DenseVectorB,
                                                d_dataValC);
#ifdef PROFILE
        time = timer.tick();
        std::cout << "Cuda Compute: " << time << std::endl;
        timer.tick();
#endif

        h_dataValC = (double *)malloc(dataValC_size);
        hipMemcpy(h_dataValC, d_dataValC, dataValC_size, hipMemcpyDeviceToHost);
        CSRMatrix<double> *ret = new CSRMatrix<double>(A->rows, DenseVectorB->cols, h_dataValC);

        hipFree(d_rowPtrA);
        hipFree(d_dataColA);
        hipFree(d_dataValA);
        hipFree(VectorOnGpu);
        hipFree(d_dataValC);
#ifdef PROFILE
        time = timer.tick();
        std::cout << "Cuda Teardown: " << time << std::endl;
#endif

        return ret;
    }

}
