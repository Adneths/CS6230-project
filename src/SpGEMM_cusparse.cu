#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""
#include <hipsparse.h>

#include "SpGEMM_cusparse.h"

#define RED     0
#define YELLOW  1
#define GREEN   2
#define AQUA    3
#define BLUE    4
#define PURPLE  5
#define BLACK   6
#define GRAY    7
#define WHITE   8

#ifdef PROFILE
#include <nvtx3/nvToolsExtCuda.h>
const uint32_t colors[] = { 0xffff0000, 0xffffff00, 0xff00ff00, 0xff00ffff, 0xff0000ff, 0xffff00ff, 0xff000000, 0xff808080, 0xffffffff };
const int num_colors = sizeof(colors)/sizeof(uint32_t);

#define POP_RANGE() nvtxRangePop();
#define NAME_THREAD(name) nvtxNameOsThread(pthread_self(), name);
#define PUSH_RANGE(name,cid) { \
    int color_id = cid; \
    color_id = color_id%num_colors;\
    nvtxEventAttributes_t eventAttrib = {0}; \
    eventAttrib.version = NVTX_VERSION; \
    eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE; \
    eventAttrib.colorType = NVTX_COLOR_ARGB; \
    eventAttrib.color = colors[color_id]; \
    eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
    eventAttrib.message.ascii = name; \
    nvtxRangePushEx(&eventAttrib); \
}
#else
#define POP_RANGE()
#define NAME_THREAD(name)
#define PUSH_RANGE(name,cid)
#endif

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return nullptr;                                                        \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return nullptr;                                                        \
    }                                                                          \
}

namespace cusparse {

CSRMatrix<double>* spgemm(CSRMatrix<double>* A, CSRMatrix<double>* B) {
    hipSetDevice(0);
//#ifdef PROFILE
    NAME_THREAD("MAIN_THREAD");
    PUSH_RANGE("CuSparse_spgemm", BLACK);
    PUSH_RANGE("CuSparse_spgemm_cudamalloc", RED);
//#endif
    double              alpha       = 1.0f;
    double              beta        = 0.0f;
    hipsparseOperation_t opA         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipDataType        computeType = HIP_R_64F;

    // Allocate
    int *d_rowPtrA, *d_dataColA, *d_rowPtrB, *d_dataColB, *d_rowPtrC, *d_dataColC;
    double *d_dataValA, *d_dataValB, *d_dataValC;
    size_t
    rowPtrA_size  = (A->rows+1) * sizeof(int),
    dataColA_size = (A->nnz) * sizeof(int),
    dataValA_size = (A->nnz) * sizeof(double),
    rowPtrB_size  = (B->rows+1) * sizeof(int),
    dataColB_size = (B->nnz) * sizeof(int),
    dataValB_size = (B->nnz) * sizeof(double);
    
    hipMalloc(&d_rowPtrA , rowPtrA_size );
    hipMalloc(&d_dataColA, dataColA_size);
    hipMalloc(&d_dataValA, dataValA_size);
    hipMalloc(&d_rowPtrB , rowPtrB_size );
    hipMalloc(&d_dataColB, dataColB_size);
    hipMalloc(&d_dataValB, dataValB_size);
    hipMalloc(&d_rowPtrC , rowPtrA_size );

//#ifdef PROFILE
    POP_RANGE();
    PUSH_RANGE("CuSparse_spgemm_HtoD", AQUA);
//#endif
    // Copy
    hipMemcpy(d_rowPtrA , A->rowPtr , rowPtrA_size , hipMemcpyHostToDevice);
    hipMemcpy(d_dataColA, A->dataCol, dataColA_size, hipMemcpyHostToDevice);
    hipMemcpy(d_dataValA, A->dataVal, dataValA_size, hipMemcpyHostToDevice);
    hipMemcpy(d_rowPtrB , B->rowPtr , rowPtrB_size , hipMemcpyHostToDevice);
    hipMemcpy(d_dataColB, B->dataCol, dataColB_size, hipMemcpyHostToDevice);
    hipMemcpy(d_dataValB, B->dataVal, dataValB_size, hipMemcpyHostToDevice);
    //--------------------------------------------------------------------------
    // Cusparse Setup
//#ifdef PROFILE
    POP_RANGE();
    PUSH_RANGE("CuSparse_spgemm_setupcusparse", PURPLE);
//#endif
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA, matB, matC;
    void*  dBuffer1    = NULL, *dBuffer2   = NULL;
    size_t bufferSize1 = 0,    bufferSize2 = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A->rows, A->cols, A->nnz,
                                      d_rowPtrA, d_dataColA, d_dataValA,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matB, B->rows, B->cols, B->nnz,
                                      d_rowPtrB, d_dataColB, d_dataValB,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matC, A->rows, B->cols, 0,
                                      NULL, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) )
    hipsparseSpGEMMDescr_t spgemmDesc;
    CHECK_CUSPARSE( hipsparseSpGEMM_createDescr(&spgemmDesc) )
    //--------------------------------------------------------------------------

//#ifdef PROFILE
    POP_RANGE();
    PUSH_RANGE("CuSparse_spgemm_compute", GREEN);
//#endif
    // SpGEMM Computation
    // ask bufferSize1 bytes for external memory
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, NULL) )
    CHECK_CUDA( hipMalloc((void**) &dBuffer1, bufferSize1) )
    // inspect the matrices A and B to understand the memory requirement for
    // the next step
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, dBuffer1) )

    // ask bufferSize2 bytes for external memory
    CHECK_CUSPARSE(
        hipsparseSpGEMM_compute(handle, opA, opB,
                               &alpha, matA, matB, &beta, matC,
                               computeType, HIPSPARSE_SPGEMM_DEFAULT,
                               spgemmDesc, &bufferSize2, NULL) )
    CHECK_CUDA( hipMalloc((void**) &dBuffer2, bufferSize2) )

    // compute the intermediate product of A * B
    CHECK_CUSPARSE( hipsparseSpGEMM_compute(handle, opA, opB,
                                           &alpha, matA, matB, &beta, matC,
                                           computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDesc, &bufferSize2, dBuffer2) )

    // get matrix C non-zero entries C_nnz
    int64_t C_num_rows1, C_num_cols1, C_nnz;
    CHECK_CUSPARSE( hipsparseSpMatGetSize(matC, &C_num_rows1, &C_num_cols1, &C_nnz) )
    // allocate matrix C
//#ifdef PROFILE
    POP_RANGE();
    PUSH_RANGE("CuSparse_spgemm_cudamalloc", RED);
//#endif
    CHECK_CUDA( hipMalloc((void**) &d_dataColC, C_nnz * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &d_dataValC,  C_nnz * sizeof(double)) )

//#ifdef PROFILE
    POP_RANGE();
    PUSH_RANGE("CuSparse_spgemm_DtoD", AQUA);
//#endif
    // NOTE: if 'beta' != 0, the values of C must be update after the allocation
    //       of dC_values, and before the call of hipsparseSpGEMM_copy
    // update matC with the new pointers
    CHECK_CUSPARSE( hipsparseCsrSetPointers(matC, d_rowPtrC, d_dataColC, d_dataValC) )

    // if beta != 0, hipsparseSpGEMM_copy reuses/updates the values of dC_values
    // copy the final products to the matrix C
    CHECK_CUSPARSE(
        hipsparseSpGEMM_copy(handle, opA, opB,
                            &alpha, matA, matB, &beta, matC,
                            computeType, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc) )

//#ifdef PROFILE
    POP_RANGE();
    PUSH_RANGE("CuSparse_spgemm_destroycusparse", PURPLE);
//#endif
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseSpGEMM_destroyDescr(spgemmDesc) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    
//#ifdef PROFILE
    POP_RANGE();
    PUSH_RANGE("CuSparse_spgemm_malloc", RED);
//#endif
    int *h_rowPtrC, *h_dataColC;
    double *h_dataValC;
    h_rowPtrC = (int*)malloc((A->rows+1) * sizeof(int));
    h_dataColC = (int*)malloc(C_nnz * sizeof(int));
    h_dataValC = (double*)malloc(C_nnz * sizeof(double));

//#ifdef PROFILE
    POP_RANGE();
    PUSH_RANGE("CuSparse_spgemm_DtoH", AQUA);
//#endif
    hipMemcpy(h_rowPtrC, d_rowPtrC, (A->rows+1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_dataColC, d_dataColC, C_nnz * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_dataValC, d_dataValC, C_nnz * sizeof(double), hipMemcpyDeviceToHost);

//#ifdef PROFILE
    POP_RANGE();
    PUSH_RANGE("CuSparse_spgemm_reconstruct", GRAY);
//#endif
    CSRMatrix<double>* ret = new CSRMatrix<double>(A->rows, B->cols, h_rowPtrC, h_dataColC, h_dataValC, C_nnz);

//#ifdef PROFILE
    POP_RANGE();
    PUSH_RANGE("CuSparse_spgemm_cudafree", BLUE);
//#endif
    hipFree(dBuffer1);
    hipFree(dBuffer2);
    hipFree(d_rowPtrA );
    hipFree(d_dataColA);
    hipFree(d_dataValA);
    hipFree(d_rowPtrB );
    hipFree(d_dataColB);
    hipFree(d_dataValB);
    hipFree(d_rowPtrC );
    hipFree(d_dataColC);
    hipFree(d_dataValC);
//#ifdef PROFILE
    POP_RANGE();
    POP_RANGE();
//#endif
    
    return ret;
}
}
