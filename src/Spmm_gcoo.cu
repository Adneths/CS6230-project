#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""
#include <stdio.h>
#include "Spmm_gcoo.h"

#ifdef PROFILE
#include "timer.h"
#endif

#define CHECK_CUDA(func)                                               \
    {                                                                  \
        hipError_t status = (func);                                   \
        if (status != hipSuccess)                                     \
        {                                                              \
            printf("CUDA API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipGetErrorString(status), status);      \
            return nullptr;                                            \
        }                                                              \
    }

namespace GCOOSPMM
{
    // CUDA kernel function
    // *a thread block is responsible for a group calculation
    __global__ void GCOOSpMMKernel(double *values, int *cols, int *rows, int *gIdxes,
                                   int *nnzPerGroup, int wA, int hA, double *B, int wB, int hB, double *C)
    {
        // find the location of the thread in C's column
        int Cj = blockIdx.y * b_value + threadIdx.x;
        // find the "start" location of the thread in C's row
        int Ci0 = blockIdx.x * p_value;
        // a local array for storing the submatrix of C calculated by a thread
        double c[p_value] = {0};

        int CurGroupNNZ = nnzPerGroup[blockIdx.x];

        // find the "start" index of current group
        double *vals = values + gIdxes[blockIdx.x];
        int *co_cols = cols + gIdxes[blockIdx.x];
        int *co_rows = rows + gIdxes[blockIdx.x];

        // calculate the iteration of current group, b also means the number of thread in a thread block
        int iter = (CurGroupNNZ + b_value - 1) / b_value; // inorder to move all the data of a thread block into the shared memory

        // 遍历当前组中的所有非零元素
        for (int i = 0; i < iter; ++i)
        {
            int cooOffset = i * b_value; // 计算COO偏移量

            __shared__ double sVals[b_value];
            __shared__ int sCols[b_value];
            __shared__ int sRows[b_value];

            // 加载当前组的COO数据到共享内存
            if (threadIdx.x < b_value && (cooOffset + threadIdx.x) < CurGroupNNZ)
            { // 确保不会超出当前组的大小
                sVals[threadIdx.x] = vals[cooOffset + threadIdx.x];
                sCols[threadIdx.x] = co_cols[cooOffset + threadIdx.x];
                sRows[threadIdx.x] = co_rows[cooOffset + threadIdx.x];
            }
            else
            {
                // 超出部分的线程不加载任何数据
                sVals[threadIdx.x] = 0;
                sCols[threadIdx.x] = -1; // 使用-1来标记无效的索引
                sRows[threadIdx.x] = -1;
            }
            __syncthreads(); // 确保所有线程都加载完数据

            // 如果当前线程对应的列索引在B矩阵范围内，则执行计算
            if (Cj < wB)
            {
                for (int j = 0; j < b_value && sCols[j] != -1; ++j)
                {
                    int col = sCols[j];
                    int row = sRows[j];
                    double av = sVals[j];
                    if (col == -1)
                        continue; // 跳过无效数据
                    double bv = B[col * wB + Cj];
                    int outIdx = row % p_value;
                    c[outIdx] += av * bv; // 执行乘加操作

                    // 内部循环，用于寻找可以重用bv值的连续非零元素
                    int k = 1;
                    while (j + k < b_value && sCols[j + k] == col)
                    {
                        if (j + k >= CurGroupNNZ)
                            break;
                        av = sVals[j + k];
                        row = sRows[j + k];
                        outIdx = row % p_value;
                        c[outIdx] += av * bv;
                        k++;
                    }
                    j += k - 1; // 跳过已经处理的元素
                }
            }
            __syncthreads(); // 等待所有线程完成这一轮计算
        }

        // 将中间结果数组c中的值写回到全局内存的C矩阵
        if (Cj < wB)
        {
            for (int i = 0; i < p_value; ++i)
            {
                C[(Ci0 + i) * wB + Cj] = c[i];
                printf("c[%d]=%f\n", i, c[i]);
            }
        }
    }

    CSRMatrix<double> *spmm(GCOO<double> *A, dense_mat<double> *DenseMatrixB)
    {
#ifdef PROFILE
        Timer timer;
        auto time = timer.tick();
#endif

        double *d_Avalues, *d_Bvalues, *d_Cvalues, *h_Cvalues;
        int *d_Acols, *d_Arows, *d_AgIdxes, *d_AnnzPerGroup;

        size_t
            rowPtrA_size = (A->nnz) * sizeof(int),
            colPtrA_size = (A->nnz) * sizeof(int),
            dataValA_size = (A->nnz) * sizeof(double),
            gIdxesA_size = (A->num_group) * sizeof(int),
            nnzPerGroupA_size = (A->num_group) * sizeof(int),
            DenseMatrixB_size = (DenseMatrixB->total_size) * sizeof(double),
            dataValC_size = (A->num_row * DenseMatrixB->col_num) * sizeof(double);

        CHECK_CUDA(hipMalloc(&d_Arows, rowPtrA_size));
        CHECK_CUDA(hipMalloc(&d_Acols, colPtrA_size));
        CHECK_CUDA(hipMalloc(&d_Avalues, dataValA_size));
        CHECK_CUDA(hipMalloc(&d_AgIdxes, gIdxesA_size));
        CHECK_CUDA(hipMalloc(&d_AnnzPerGroup, nnzPerGroupA_size));
        CHECK_CUDA(hipMalloc(&d_Bvalues, DenseMatrixB_size));
        CHECK_CUDA(hipMalloc(&d_Cvalues, dataValC_size));

        hipMemcpy(d_Arows, A->rows, rowPtrA_size, hipMemcpyHostToDevice);
        hipMemcpy(d_Acols, A->cols, colPtrA_size, hipMemcpyHostToDevice);
        hipMemcpy(d_Avalues, A->values, dataValA_size, hipMemcpyHostToDevice);
        hipMemcpy(d_AgIdxes, A->gIdexs, gIdxesA_size, hipMemcpyHostToDevice);
        hipMemcpy(d_AnnzPerGroup, A->nnzpergroup, nnzPerGroupA_size, hipMemcpyHostToDevice);
        hipMemcpy(d_Bvalues, DenseMatrixB->matrix, DenseMatrixB_size, hipMemcpyHostToDevice);
        hipMemset(d_Cvalues, 0, dataValC_size);

        // dim3 threadsPerBlock(b_value);
        dim3 threadsPerBlock(64);
        dim3 numBlocks((A->num_row + p_value - 1) / p_value, (DenseMatrixB->col_num + b_value - 1) / b_value);
#ifdef PROFILE
        time = timer.tick();
        std::cout << "Cuda Setup: " << time << std::endl;
        timer.tick();
#endif

        GCOOSpMMKernel<<<numBlocks, threadsPerBlock>>>(d_Avalues, d_Acols, d_Arows, d_AgIdxes, d_AnnzPerGroup,
                                                       A->num_col, A->num_row, d_Bvalues, DenseMatrixB->col_num,
                                                       DenseMatrixB->row_num, d_Cvalues);
        hipDeviceSynchronize();
#ifdef PROFILE
        time = timer.tick();
        std::cout << "Cuda Compute: " << time << std::endl;
        timer.tick();
#endif

        h_Cvalues = (double *)malloc(dataValC_size);
        hipMemcpy(h_Cvalues, d_Cvalues, dataValC_size, hipMemcpyDeviceToHost);
        CSRMatrix<double> *ret = new CSRMatrix<double>(A->num_row, DenseMatrixB->col_num, h_Cvalues);

        hipFree(d_Avalues);
        hipFree(d_Bvalues);
        hipFree(d_Cvalues);
        hipFree(d_Acols);
        hipFree(d_Arows);
        hipFree(d_AgIdxes);
        hipFree(d_AnnzPerGroup);

#ifdef PROFILE
        time = timer.tick();
        std::cout << "Cuda Teardown: " << time << std::endl;
#endif

        return ret;
    }

}
