
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""

#include "TileSpGEMM_cuda.h"

#ifdef PROFILE
#include <nvtx3/nvToolsExtCuda.h>
#endif

namespace cuda {

CSRMatrix<double>* tile_spgemm(CSRMatrix<double>* A, CSRMatrix<double>* B) {
    TileSpMatrix<double>* tileA = new TileSpMatrix<double>(A);
    std::cout << tileA << std::endl;
    return nullptr;
}

}