
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""

#include "TileSpGEMM_cuda.h"

#ifdef PROFILE
#include <nvtx3/nvToolsExtCuda.h>
#endif

namespace cuda {

CSRMatrix<double>* tile_spgemm(CSRMatrix<double>* A, CSRMatrix<double>* B) {
    TileSpMatrix<double>* tileA = new TileSpMatrix<double>(A);
    TileSpMatrix<double>* tileB = new TileSpMatrix<double>(B);
    std::cout << tileA << std::endl;
    return nullptr;
}

}

namespace cusparse {
    CSRMatrix<double>* spgemm_symbolic(CSRMatrix<double>* A, CSRMatrix<double>* B) {
#ifdef PROFILE
    nvtxNameOsThread(pthread_self(), "MAIN_THREAD");
    nvtxRangePushA("CuSparse_spgemm");
    nvtxRangePushA("CuSparse_spgemm_setup");
#endif
    double              alpha       = 1.0f;
    double              beta        = 0.0f;
    hipsparseOperation_t opA         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipDataType        computeType = HIP_R_64F;

    // Allocate
    int *d_rowPtrA, *d_dataColA, *d_rowPtrB, *d_dataColB, *d_rowPtrC, *d_dataColC;
    double *d_dataValA, *d_dataValB, *d_dataValC;
    size_t
    rowPtrA_size  = (A->rows+1) * sizeof(int),
    dataColA_size = (A->nnz) * sizeof(int),
    dataValA_size = (A->nnz) * sizeof(double),
    rowPtrB_size  = (B->rows+1) * sizeof(int),
    dataColB_size = (B->nnz) * sizeof(int),
    dataValB_size = (B->nnz) * sizeof(double);
    
    hipMalloc(&d_rowPtrA , rowPtrA_size );
    hipMalloc(&d_dataColA, dataColA_size);
    hipMalloc(&d_dataValA, dataValA_size);
    hipMalloc(&d_rowPtrB , rowPtrB_size );
    hipMalloc(&d_dataColB, dataColB_size);
    hipMalloc(&d_dataValB, dataValB_size);
    hipMalloc(&d_rowPtrC , rowPtrA_size );

    // Copy
    hipMemcpy(d_rowPtrA , A->rowPtr , rowPtrA_size , hipMemcpyHostToDevice);
    hipMemcpy(d_dataColA, A->dataCol, dataColA_size, hipMemcpyHostToDevice);
    hipMemcpy(d_dataValA, A->dataVal, dataValA_size, hipMemcpyHostToDevice);
    hipMemcpy(d_rowPtrB , B->rowPtr , rowPtrB_size , hipMemcpyHostToDevice);
    hipMemcpy(d_dataColB, B->dataCol, dataColB_size, hipMemcpyHostToDevice);
    hipMemcpy(d_dataValB, B->dataVal, dataValB_size, hipMemcpyHostToDevice);
    //--------------------------------------------------------------------------
    // Cusparse Setup
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA, matB, matC;
    void*  dBuffer1    = NULL, *dBuffer2   = NULL;
    size_t bufferSize1 = 0,    bufferSize2 = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A->rows, A->cols, A->nnz,
                                      d_rowPtrA, d_dataColA, d_dataValA,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matB, B->rows, B->cols, B->nnz,
                                      d_rowPtrB, d_dataColB, d_dataValB,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matC, A->rows, B->cols, 0,
                                      NULL, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) )
    hipsparseSpGEMMDescr_t spgemmDesc;
    CHECK_CUSPARSE( hipsparseSpGEMM_createDescr(&spgemmDesc) )
    //--------------------------------------------------------------------------

#ifdef PROFILE
    nvtxRangePop();
    nvtxRangePushA("CuSparse_spgemm_compute");
#endif
    // SpGEMM Computation
    // ask bufferSize1 bytes for external memory
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, NULL) )
    CHECK_CUDA( hipMalloc((void**) &dBuffer1, bufferSize1) )
    // inspect the matrices A and B to understand the memory requirement for
    // the next step
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, dBuffer1) )

    // ask bufferSize2 bytes for external memory
    CHECK_CUSPARSE(
        hipsparseSpGEMM_compute(handle, opA, opB,
                               &alpha, matA, matB, &beta, matC,
                               computeType, HIPSPARSE_SPGEMM_DEFAULT,
                               spgemmDesc, &bufferSize2, NULL) )
    CHECK_CUDA( hipMalloc((void**) &dBuffer2, bufferSize2) )

    // compute the intermediate product of A * B
    CHECK_CUSPARSE( hipsparseSpGEMM_compute(handle, opA, opB,
                                           &alpha, matA, matB, &beta, matC,
                                           computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDesc, &bufferSize2, dBuffer2) )
#ifdef PROFILE
    nvtxRangePop();
    nvtxRangePushA("CuSparse_spgemm_deardown");
#endif
    // get matrix C non-zero entries C_nnz
    int64_t C_num_rows1, C_num_cols1, C_nnz;
    CHECK_CUSPARSE( hipsparseSpMatGetSize(matC, &C_num_rows1, &C_num_cols1, &C_nnz) )
    // allocate matrix C
    CHECK_CUDA( hipMalloc((void**) &d_dataColC, C_nnz * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &d_dataValC,  C_nnz * sizeof(double)) )

    // NOTE: if 'beta' != 0, the values of C must be update after the allocation
    //       of dC_values, and before the call of hipsparseSpGEMM_copy
    // update matC with the new pointers
    CHECK_CUSPARSE( hipsparseCsrSetPointers(matC, d_rowPtrC, d_dataColC, d_dataValC) )

    // if beta != 0, hipsparseSpGEMM_copy reuses/updates the values of dC_values
    // copy the final products to the matrix C
    CHECK_CUSPARSE(
        hipsparseSpGEMM_copy(handle, opA, opB,
                            &alpha, matA, matB, &beta, matC,
                            computeType, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc) )

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseSpGEMM_destroyDescr(spgemmDesc) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    
    int *h_rowPtrC, *h_dataColC;
    double *h_dataValC;
    h_rowPtrC = (int*)malloc((A->rows+1) * sizeof(int));
    h_dataColC = (int*)malloc(C_nnz * sizeof(int));
    h_dataValC = (double*)malloc(C_nnz * sizeof(double));

    hipMemcpy(h_rowPtrC, d_rowPtrC, (A->rows+1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_dataColC, d_dataColC, C_nnz * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_dataValC, d_dataValC, C_nnz * sizeof(double), hipMemcpyDeviceToHost);

    CSRMatrix<double>* ret = new CSRMatrix<double>(A->rows, B->cols, h_rowPtrC, h_dataColC, h_dataValC, C_nnz);

    hipFree(dBuffer1);
    hipFree(dBuffer2);
    hipFree(d_rowPtrA );
    hipFree(d_dataColA);
    hipFree(d_dataValA);
    hipFree(d_rowPtrB );
    hipFree(d_dataColB);
    hipFree(d_dataValB);
    hipFree(d_dataValC);
#ifdef PROFILE
    nvtxRangePop();
    nvtxRangePop();
#endif
    
    return ret;
}
}