#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""
#include <hipsparse.h>

#include "SpMM_cusparse.h"

#ifdef PROFILE
#include "timer.h"
#endif

#define CHECK_CUDA(func)                                               \
    {                                                                  \
        hipError_t status = (func);                                   \
        if (status != hipSuccess)                                     \
        {                                                              \
            printf("CUDA API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipGetErrorString(status), status);      \
            return nullptr;                                            \
        }                                                              \
    }

#define CHECK_CUSPARSE(func)                                               \
    {                                                                      \
        hipsparseStatus_t status = (func);                                  \
        if (status != HIPSPARSE_STATUS_SUCCESS)                             \
        {                                                                  \
            printf("CUSPARSE API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipsparseGetErrorString(status), status);      \
            return nullptr;                                                \
        }                                                                  \
    }

namespace cusparse
{

    CSRMatrix<double> *spmm(CSRMatrix<double> *A, dense_mat<double> *B)
    {
#ifdef PROFILE
        Timer timer;
        auto time = timer.tick();
#endif
        double alpha = 1.0f;
        double beta = 0.0f;
        // hipsparseOperation_t opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        // hipsparseOperation_t opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        // hipDataType computeType = HIP_R_64F;

        // Allocate
        int *d_rowPtrA, *d_dataColA;
        double *d_dataValA, *d_B, *d_C;
        size_t
            rowPtrA_size = (A->rows + 1) * sizeof(int),
            dataColA_size = (A->nnz) * sizeof(int),
            dataValA_size = (A->nnz) * sizeof(double),
            B_size = (B->total_size) * sizeof(double),
            C_size = (A->rows * B->col_num) * sizeof(double);

        hipMalloc(&d_rowPtrA, rowPtrA_size);
        hipMalloc(&d_dataColA, dataColA_size);
        hipMalloc(&d_dataValA, dataValA_size);
        hipMalloc(&d_B, B_size);
        hipMalloc(&d_C, C_size);

        // Copy
        hipMemcpy(d_rowPtrA, A->rowPtr, rowPtrA_size, hipMemcpyHostToDevice);
        hipMemcpy(d_dataColA, A->dataCol, dataColA_size, hipMemcpyHostToDevice);
        hipMemcpy(d_dataValA, A->dataVal, dataValA_size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, B->matrix, B_size, hipMemcpyHostToDevice);
        //--------------------------------------------------------------------------
        // Cusparse Setup
        hipsparseHandle_t handle = NULL;
        hipsparseSpMatDescr_t matA;
        hipsparseDnMatDescr_t matB, matC;
        void *dBuffer = NULL;
        size_t bufferSize = 0;
        CHECK_CUSPARSE(hipsparseCreate(&handle))
        CHECK_CUSPARSE(hipsparseCreateCsr(&matA, A->rows, A->cols, A->nnz,
                                         d_rowPtrA, d_dataColA, d_dataValA,
                                         HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                         HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F))
        // Create dense matrix B
        CHECK_CUSPARSE(hipsparseCreateDnMat(&matB, B->row_num, B->col_num, B->col_num, d_B,
                                           HIP_R_64F, HIPSPARSE_ORDER_ROW))
        // Create dense matrix C
        CHECK_CUSPARSE(hipsparseCreateDnMat(&matC, A->rows, B->col_num, A->rows, d_C,
                                           HIP_R_64F, HIPSPARSE_ORDER_ROW))
        //--------------------------------------------------------------------------
#ifdef PROFILE
        time = timer.tick();
        std::cout << "CuSparse Setup: " << time << std::endl;
        timer.tick();
#endif
        // SpMM Computation
        // allocate an external buffer if needed
        CHECK_CUSPARSE(hipsparseSpMM_bufferSize(
            handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA, matB, &beta, matC, HIP_R_64F,
            HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize))
        CHECK_CUDA(hipMalloc(&dBuffer, bufferSize))

        // execute SpMM
        CHECK_CUSPARSE(hipsparseSpMM(handle,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha, matA, matB, &beta, matC, HIP_R_64F,
                                    HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer))
#ifdef PROFILE
        time = timer.tick();
        std::cout << "CuSparse Compute: " << time << std::endl;
        timer.tick();
#endif
        // destroy matrix/vector descriptors
        CHECK_CUSPARSE(hipsparseDestroySpMat(matA))
        CHECK_CUSPARSE(hipsparseDestroyDnMat(matB))
        CHECK_CUSPARSE(hipsparseDestroyDnMat(matC))
        CHECK_CUSPARSE(hipsparseDestroy(handle))
        //--------------------------------------------------------------------------

        double *h_C;
        h_C = (double *)malloc(A->rows * B->col_num * sizeof(double));

        hipMemcpy(h_C, d_C, A->rows * B->col_num * sizeof(double), hipMemcpyDeviceToHost);

        CSRMatrix<double> *ret = new CSRMatrix<double>(A->rows, B->col_num, h_C);

        hipFree(dBuffer);
        hipFree(d_rowPtrA);
        hipFree(d_dataColA);
        hipFree(d_dataValA);
        hipFree(d_B);
        hipFree(d_C);
#ifdef PROFILE
        time = timer.tick();
        std::cout << "CuSparse Teardown: " << time << std::endl;
#endif

        return ret;
    }
}